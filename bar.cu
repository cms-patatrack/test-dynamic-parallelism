#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cudaCheck.h"

__global__
void bar() {
  printf("bar\n");
}

#ifdef MAY_CRASH
__global__
void crash() {
  bar<<<1,1>>>();
  hipDeviceSynchronize();
}
#endif  // MAY_CRASH

void wrapper() {
  bar<<<1,1>>>();
  cudaCheck(hipGetLastError());
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
}

struct Me {

  Me() {
   std::cout << "Loaded" << std::endl;
   wrapper();
  }

};

Me me;

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdio>

#include "cudaCheck.h"

__global__ 
void foo() {
   printf("Hello\n");
}

void fooWrapper() {
  foo<<<1,1>>>();
  cudaCheck(hipGetLastError());
  cudaCheck(hipDeviceSynchronize());
}

void doCheck() {
  cudaCheck(hipDeviceSynchronize());
  cudaCheck(hipGetLastError());
}
